#include "hip/hip_runtime.h"

#include <stdio.h>

//for random intialize
#include <stdlib.h>
#include <time.h>

//for memset
#include <cstring>

#ifndef MAX_SIZE
    #define MAX_SIZE 100000000
#endif

#ifndef MIN_SIZE
    #define MIN_SIZE 1000
#endif

#ifndef BLOCK_SIZE
    #define BLOCK_SIZE 128
#endif

__global__ void mul_vec_gpu(int * a, int * b, int* c, int size)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;

	if (index < size)
		c[index] = a[index] * b[index];
}

int main()
{
    printf("Array Size, GPU Kernel Time, Host to Device Time, Device to Host Time, Total GPU Time\n");
    for (int size = MIN_SIZE; size <= MAX_SIZE; size *= 10) {
        int block_size = BLOCK_SIZE;
        hipError_t error;

        //number of bytes needed to hold element count
        size_t NO_BYTES = size * sizeof(int);

        // host pointers
        int *h_a, *h_b, *gpu_result;

        //allocate memory for host size pointers
        h_a = (int *)malloc(NO_BYTES);
        h_b = (int *)malloc(NO_BYTES);
        gpu_result = (int *)malloc(NO_BYTES);

        //initialize h_a and h_b vectors randomly
        time_t t;
        srand((unsigned)time(&t));

        for (size_t i = 0; i < size; i++)
        {
            h_a[i] = (int)(rand() & 0xFF);
            
        }

        for (size_t i = 0; i < size; i++)
        {
            h_b[i] = (int)(rand() & 0xFF);
        
        }

        memset(gpu_result, 0, NO_BYTES);

        int *d_a, *d_b, *d_c;
        hipMalloc((int **)&d_a, NO_BYTES);
        hipMalloc((int **)&d_b, NO_BYTES);
        hipMalloc((int **)&d_c, NO_BYTES);

        //kernel launch parameters
        dim3 block(block_size);
        dim3 grid((size / block.x) + 1);

        clock_t mem_htod_start, mem_htod_end;
        mem_htod_start = clock();
        hipMemcpy(d_a, h_a, NO_BYTES, hipMemcpyHostToDevice);
        hipMemcpy(d_b, h_b, NO_BYTES, hipMemcpyHostToDevice);
        mem_htod_end = clock();

        //execution time measuring in GPU
        clock_t gpu_start, gpu_end;
        gpu_start = clock();

        mul_vec_gpu<<<grid, block>>>(d_a, d_b, d_c, size);
        hipDeviceSynchronize();
        gpu_end = clock();

        clock_t mem_dtoh_start, mem_dtoh_end;
        mem_dtoh_start = clock();
        hipMemcpy(gpu_result, d_c, NO_BYTES, hipMemcpyDeviceToHost);
        mem_dtoh_end = clock();

        printf("%d,", size);

        printf("%4.6f,",
            (double)((double)(gpu_end - gpu_start) / CLOCKS_PER_SEC));

        printf("%4.6f,",
            (double)((double)(mem_htod_end - mem_htod_start) / CLOCKS_PER_SEC));

        printf("%4.6f,",
            (double)((double)(mem_dtoh_end - mem_dtoh_start) / CLOCKS_PER_SEC));

        printf("%4.6f",
            (double)((double)((mem_htod_end - mem_htod_start)
                + (gpu_end - gpu_start)
                + (mem_dtoh_end - mem_dtoh_start)) / CLOCKS_PER_SEC));

        hipFree(d_c);
        hipFree(d_b);
        hipFree(d_a);	
                
        free(gpu_result);
        free(h_a);
        free(h_b);
                
        hipDeviceReset();
        printf("\n");
    }
	return 0;
}
